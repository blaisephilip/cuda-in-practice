
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
    // Get the thread ID
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Make sure we don't go out of bounds
    if (index < n)
        y[index] = x[index] + y[index];
}

int main(void)
{
    int N = 1<<20; // 1M elements
    
    // Allocate host memory
    float *x = new float[N];
    float *y = new float[N];

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Allocate device memory
    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    // Copy arrays to device
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, d_x, d_y);

    // Copy result back to host
    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_x);
    hipFree(d_y);
    delete [] x;
    delete [] y;

    return 0;
}